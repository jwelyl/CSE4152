#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#define	IN
#define OUT
#define INOUT
#define EPSILON    0.01

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define N_EQUATIONS (1 << 26)												// ������ ����
//#define N_EQUATIONS (1 << 6)

#define BLOCK_SIZE (1 << 6)													// CUDA Ŀ�� thread block ������

//#define BLOCK_SIZE (1 << 4)

#define BLOCK_WIDTH (1 << 3)
//#define BLOCK_WIDTH (1 << 2)

#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
float* h_A;             //  2���� ���
float* h_B;             //  1���� ���
float* h_C;             //  �����
float* h_X0;            //  �� �Ǳ� �� ���� �� x0 (CPU ��갪)
float* h_X1;            //  �� �Ǳ� �� ū �� x1 (")
float* h_FX0;           //  ax0^2 + bx0 + c�� �� (")
float* h_FX1;           //  ax1^2 + bx1 + c�� �� (")
float* d_X0_GPU;        //  �� �Ǳ� �� ���� �� x0 (GPU ��갪)
float* d_X1_GPU;        //  �� �Ǳ� �� ū �� x1 (")
float* d_FX0_GPU;       //  ax0^2 + bx0 + c�� �� (")
float* d_FX1_GPU;       //  ax1^2 + bx1 + c�� �� (")


hipError_t find_roots_GPU(IN float* A_CPU, IN float* B_CPU, IN float* C_CPU,
    OUT float* X0_GPU, OUT float* X1_GPU, OUT float* FX0_GPU, OUT float* FX1_GPU, int n);

__global__ void find_roots_Kernel(IN float* d_A, IN float* d_B, IN float* d_C,
    OUT float* d_X0, OUT float* d_X1, OUT float* d_FX0, OUT float* d_FX1, int n) {
    /*Todo*/
    //printf("GPU Kernel code!!\n");

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int id = gridDim.x * blockDim.x * row + col;

    float a, b, c, d, x0, x1, tmp;

    a = d_A[id];    b = d_B[id];    c = d_C[id];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0 * a);
    d_X0[id] = x0 = (-b - d) * tmp;
    d_X1[id] = x1 = (-b + d) * tmp;
    d_FX0[id] = (a * x0 + b) * x0 + c;
    d_FX1[id] = (a * x1 + b) * x1 + c;
}

void find_roots_CPU(IN float* A_CPU, IN float* B_CPU, IN float* C_CPU, 
    OUT float* X0_CPU, OUT float* X1_CPU, OUT float* FX0_CPU, OUT float* FX1_CPU, int n) {
    int i;

    float a, b, c, d, x0, x1, tmp;

    //printf("CPU roots\n");
    for (i = 0; i < n; i++) {
        a = A_CPU[i];   b = B_CPU[i];   c = C_CPU[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0 * a);
        X0_CPU[i] = x0 = (-b - d) * tmp;
        X1_CPU[i] = x1 = (-b + d) * tmp;

        //printf("i = %d, x0 = %f, x1 = %f\n", i, X0_CPU[i], X1_CPU[i]);

        FX0_CPU[i] = (a * x0 + b) * x0 + c;
        FX1_CPU[i] = (a * x1 + b) * x1 + c;
    }
}

void read_bin_file() {
    printf("***Binary File Read Start!!\n");
    FILE* fp_a = fopen("A.bin", "rb");
    FILE* fp_b = fopen("B.bin", "rb");
    FILE* fp_c = fopen("C.bin", "rb");
    int tmp;
    fread(&N, sizeof(int), 1, fp_a);
    tmp = N;
    fread(&N, sizeof(int), 1, fp_b);
    if (tmp != N) goto FILE_ERROR;
    fread(&N, sizeof(int), 1, fp_c);
    if (tmp != N) goto FILE_ERROR;

    h_A = (float*)malloc(N * sizeof(float));
    h_B = (float*)malloc(N * sizeof(float));
    h_C = (float*)malloc(N * sizeof(float));
    h_X0 = (float*)malloc(N * sizeof(float));
    h_X1 = (float*)malloc(N * sizeof(float)); 
    h_FX0 = (float*)malloc(N * sizeof(float));
    h_FX1 = (float*)malloc(N * sizeof(float));
    d_X0_GPU = (float*)malloc(N * sizeof(float));
    d_X1_GPU = (float*)malloc(N * sizeof(float));
    d_FX0_GPU = (float*)malloc(N * sizeof(float));
    d_FX1_GPU = (float*)malloc(N * sizeof(float));

    fread(h_A, sizeof(float), N, fp_a);
    fread(h_B, sizeof(float), N, fp_b);
    fread(h_C, sizeof(float), N, fp_c);

    //for (int i = 0; i < N; i++) {
    //    printf("equation %d : %fx^2 + %fx + %f = 0\n", i, h_A[i], h_B[i], h_C[i]);
    //}

    fclose(fp_a);
    fclose(fp_b);
    fclose(fp_c);
    printf("***Binary File Read End!!\n\n");
    return;

FILE_ERROR:
    fprintf(stderr, "init_bin_file Error!\n");
    exit(-1);
}

void write_bin_file() {
    printf("\n***Binary File Write Start!!\n");
    FILE* fp_x0 = fopen("X0.bin", "wb");
    FILE* fp_x1 = fopen("X1.bin", "wb");
    FILE* fp_fx0 = fopen("FX0.bin", "wb");
    FILE* fp_fx1 = fopen("FX1.bin", "wb");
    
    fwrite(&N, sizeof(int), 1, fp_x0);
    fwrite(&N, sizeof(int), 1, fp_x1);
    fwrite(&N, sizeof(int), 1, fp_fx0);
    fwrite(&N, sizeof(int), 1, fp_fx1);
   
    fwrite(d_X0_GPU, sizeof(float), N, fp_x0);
    fwrite(d_X1_GPU, sizeof(float), N, fp_x1);
    fwrite(d_FX0_GPU, sizeof(float), N, fp_fx0);
    fwrite(d_FX1_GPU, sizeof(float), N, fp_fx1);

    fclose(fp_x0);
    fclose(fp_x1);
    fclose(fp_fx0);
    fclose(fp_fx1);

    printf("***Binary File Write End!!\n\n");
    return;
}

void init_bin_file(IN int n) {
    printf("***Binary File Create Start!!\n");
    srand((unsigned)time(NULL));
    FILE* fp_a = fopen("A.bin", "wb");
    FILE* fp_b = fopen("B.bin", "wb");
    FILE* fp_c = fopen("C.bin", "wb");
    fwrite(&n, sizeof(int), 1, fp_a);
    fwrite(&n, sizeof(int), 1, fp_b);
    fwrite(&n, sizeof(int), 1, fp_c);

    int i;

    for (i = 0; i < n; i++) {
        float input_a, input_b, input_c;

        do {
            //printf("do_while! %d\n", i);
            input_a = 200.0f * ((float)rand() / RAND_MAX) - 100.0f;
           // printf("input_a = %f\n", input_a);
            input_b = 200.0f * ((float)rand() / RAND_MAX) - 100.0f;
           // printf("input_b = %f\n", input_b);
            input_c = 200.0f * ((float)rand() / RAND_MAX) - 100.0f;
           // printf("input_c = %f\n", input_c);
        } while ((input_b * input_b - 4.0f * input_a * input_c) <= 0.0f);
        
        //printf("%d��° ������ : %fx^2 + %fx + %f = 0\n", i, input_a, input_b, input_c);

        fwrite(&input_a, sizeof(float), 1, fp_a);
        fwrite(&input_b, sizeof(float), 1, fp_b);
        fwrite(&input_c, sizeof(float), 1, fp_c);
    }

    fclose(fp_a);
    fclose(fp_b);
    fclose(fp_c);
    printf("***Binary File Create End!!\n\n");
}

int main()
{
    int i;
    init_bin_file(N_EQUATIONS);
    read_bin_file();

    CHECK_TIME_START;
    find_roots_CPU(h_A, h_B, h_C, h_X0, h_X1, h_FX0, h_FX1, N);
    CHECK_TIME_END(compute_time);
    find_roots_GPU(h_A, h_B, h_C, d_X0_GPU, d_X1_GPU, d_FX0_GPU, d_FX1_GPU, N);
    /*
    printf("\nCPU ���\n");
    for (i = 0; i < N; i++) {
        printf("%d��° ������ CPU �� x0 = %f, x1 = %f\n", i, h_X0[i], h_X1[i]);
        printf("%d��° ������ CPU ���԰� f(x0) = %f, f(x1) = %f\n\n", i, h_FX0[i], h_FX1[i]);
    }
    printf("\nGPU ���\n");
    for (i = 0; i < N; i++) {
        printf("%d��° ������ GPU �� x0 = %f, x1 = %f\n", i, d_X0_GPU[i], d_X1_GPU[i]);
        printf("%d��° ������ GPU ���԰� f(x0) = %f, f(x1) = %f\n\n", i, d_FX0_GPU[i], d_FX1_GPU[i]);
    }
    printf("\n");
    */
    for (i = 0; i < N; i++) {
        if (fabs(h_X0[i] - d_X0_GPU[i]) > EPSILON) {
            printf("In i = %d\n", i);
            printf("x0 = %f : CPU, \t x0 = %f : GPU\n", h_X0[i], d_X0_GPU[i]);
            break;
        }
        if (fabs(h_X1[i] - d_X1_GPU[i]) > EPSILON) {
            printf("In i = %d\n", i);
            printf("x1 = %f : CPU, \t x1 = %f : GPU\n", h_X1[i], d_X1_GPU[i]);
            break;
        }
        /*
        if (fabs(h_FX0[i] - d_FX0_GPU[i]) > EPSILON) {
            printf("In i = %d\n", i);
            printf("FX0 = %f : CPU, \t FX0 = %f : GPU\n", h_FX0[i], d_FX0_GPU[i]);
            break;
        }
        if (fabs(h_FX1[i] - d_FX1_GPU[i]) > EPSILON) {
            printf("In i = %d\n", i);
            printf("FX1 = %f : CPU, \t FX1 = %f : GPU\n", h_FX1[i], d_FX1_GPU[i]);
            break;
        }
        */
    }
    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    printf("***CPU compute time : %.3f ms\n", compute_time);
    printf("***GPU compute time : %.3f ms\n", device_time);

    write_bin_file();

    //printf("\n***Binary File Write Start!!\n");
    //printf("***Binary File Write End!!\n");

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_X0);
    free(h_X1);
    free(h_FX0);
    free(h_FX1);
    free(d_X0_GPU);
    free(d_X1_GPU);
    free(d_FX0_GPU);
    free(d_FX1_GPU);

    return 0;
}

hipError_t find_roots_GPU(IN float* A_CPU, IN float* B_CPU, IN float* C_CPU,
    OUT float* X0_GPU, OUT float* X1_GPU, OUT float* FX0_GPU, OUT float* FX1_GPU, int n) {
    hipError_t cudaStatus;
    /*Todo*/
    //printf("Sum_n_elements_GPU!\n");

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipDeviceProp_t deviceProp;
    CUDA_CALL(hipGetDeviceProperties(&deviceProp, 0));

    float* d_A, * d_B, * d_C;
    float* d_X0, * d_X1, * d_FX0, * d_FX1;

    size_t size = n * sizeof(int);

    CUDA_CALL(hipMalloc(&d_A, size));
    CUDA_CALL(hipMemcpy(d_A, A_CPU, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&d_B, size));
    CUDA_CALL(hipMemcpy(d_B, B_CPU, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&d_C, size));
    CUDA_CALL(hipMemcpy(d_C, C_CPU, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&d_X0, size));
    CUDA_CALL(hipMalloc(&d_X1, size));
    CUDA_CALL(hipMalloc(&d_FX0, size));
    CUDA_CALL(hipMalloc(&d_FX1, size));

    dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 dimGrid((int)sqrt(n) / dimBlock.x, (int)sqrt(n) / dimBlock.y);

    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();
    find_roots_Kernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C, d_X0, d_X1, d_FX0, d_FX1, n);
    CHECK_TIME_END_GPU(device_time);
    CHECK_TIME_DEST_GPU();

    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(X0_GPU, d_X0, size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(X1_GPU, d_X1, size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(FX0_GPU, d_FX0, size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(FX1_GPU, d_FX1, size, hipMemcpyDeviceToHost));


Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X0);
    hipFree(d_X1);
    hipFree(d_FX0);
    hipFree(d_FX1);

    return cudaStatus;
}
